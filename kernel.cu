#include "hip/hip_runtime.h"
﻿#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <sstream>
#include <random>
#include <algorithm>

#define HASH_RANGE 23
#define HASH_RANGE_SQUARED (HASH_RANGE * HASH_RANGE)

using uint = unsigned int;

// CUDA kernel to compute the hashes
__global__ void computeHashes( int* d_primes, int primeCount, uint* d_topHashes, int* d_topPrime1 )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= primeCount ) return;

    int xorResults[HASH_RANGE_SQUARED];
    int prime1 = d_primes[i];
    uint topHashQuality = 0;
    int topPrime = 0;
    
    int halfPrimeCount = primeCount / 2;
    for( int j = 1; j <= halfPrimeCount; ++j )
    {
        int prime2 = d_primes[(i + j) % primeCount];
        uint hashQuality = 0;
        int resultIndex = 0;
        int xorBase = 0;

        for( int k = 0; k < HASH_RANGE; ++k )
        {
            int xorAdd = 0;
            for( int l = 0; l < HASH_RANGE; ++l )
            {
                xorResults[resultIndex++] = xorBase ^ xorAdd;
                xorAdd += prime2;
            }
            xorBase += prime1;
        }

        for( int m = 0; m < HASH_RANGE_SQUARED - 1; ++m )
        {
            for( int n = m + 1; n < HASH_RANGE_SQUARED; ++n )
            {
                unsigned int xorR = xorResults[m] ^ xorResults[n];
                hashQuality += __popc( xorR );
                hashQuality += __popc( xorR << 24 );
            }
        }

        if( hashQuality > topHashQuality )
        {
            topHashQuality = hashQuality;
            topPrime = prime2;
        }
    }

    d_topHashes[i] = topHashQuality;
    d_topPrime1[i] = topPrime;
}

int main()
{
    int cudeDev = 0;
    hipSetDevice(cudeDev);
    // Get and print some information about the selected device
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, cudeDev);
    std::cout << "Using CUDA Device : " << deviceProp.name << std::endl;

    std::vector<int> primes;
#if 0
    for( int fileIndex = 3; fileIndex <= 10; ++fileIndex )
    {
        std::ifstream file( "./primes" + std::to_string( fileIndex ) + ".txt" );
#else
    {
        std::ifstream file( "./top-percentile-primes.txt" );
#endif
        std::string line;
        while( std::getline( file, line ) )
        {
            size_t pos = 0;
            while( ( pos = line.find( '\t' ) ) != std::string::npos )
            {
                primes.push_back( std::stoi( line.substr( 0, pos ) ) );
                line.erase( 0, pos + 1 );
            }
            if( !line.empty() )
            {
                primes.push_back( std::stoi( line ) );
            }
        }
    }

    std::cout << "Loaded Primes: " << primes.size() << std::endl;

    // Randomly sort the primes vector
    std::random_device rd;
    std::mt19937 g(rd());
    std::shuffle(primes.begin(), primes.end(), g);

    std::cout << "Primes randomly sorted." << std::endl;

    int primeCount = primes.size();
    std::vector<uint> topHashes( primeCount, 0 );
    std::vector<int> topPrime1( primeCount, 0 );

    int* d_primes;
    uint* d_topHashes;
    int* d_topPrime1;
    hipMalloc( (void**)&d_primes, primeCount * sizeof( int ) );
    hipMalloc( (void**)&d_topHashes, primeCount * sizeof( uint ) );
    hipMalloc( (void**)&d_topPrime1, primeCount * sizeof( int ) );

    hipMemcpy( d_primes, primes.data(), primeCount * sizeof( int ), hipMemcpyHostToDevice );
    hipMemcpy( d_topHashes, topHashes.data(), primeCount * sizeof( uint ), hipMemcpyHostToDevice );
    hipMemcpy( d_topPrime1, topPrime1.data(), primeCount * sizeof( int ), hipMemcpyHostToDevice );

    float time;
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 

    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, computeHashes, 0, 0 );

    // Round up according to array size 
    gridSize = ( primeCount + blockSize - 1 ) / blockSize;

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf( "Occupancy calculator elapsed time:  %.3f s \n", time / 1000 );
    printf( "Blocksize %i\n", blockSize );

#if 1
    // Round up according to array size 
    int testCount = 4096;
    int testGridSize = ( testCount + blockSize - 1 ) / blockSize;

    hipEventRecord( start, 0 );

    computeHashes << <testGridSize, blockSize >> > ( d_primes, primeCount, d_topHashes, d_topPrime1 );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf( "Test Kernel elapsed time:  %.3f s \n", time / 1000 );
    printf( "Estimated full time:  %.3f h \n", time / (1000 * 60 * 60) * ((float)primeCount / testCount) );
#endif

    hipEventRecord( start, 0 );

    computeHashes << <gridSize, blockSize >> > ( d_primes, primeCount, d_topHashes, d_topPrime1 );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );
    printf( "Kernel elapsed time:  %.3f h \n", time / ( 1000 * 60 * 60 ) );

    hipMemcpy( topHashes.data(), d_topHashes, primeCount * sizeof( uint ), hipMemcpyDeviceToHost );
    hipMemcpy( topPrime1.data(), d_topPrime1, primeCount * sizeof( int ), hipMemcpyDeviceToHost );

    hipFree( d_primes );
    hipFree( d_topHashes );
    hipFree( d_topPrime1 );

    std::cout << "All Complete!" << std::endl;

    auto now = std::chrono::system_clock::now();
    auto now_c = std::chrono::system_clock::to_time_t( now );
    std::stringstream ss;
    ss << std::put_time( std::localtime( &now_c ), "%Y%m%d_%H%M%S" );
    std::string timestamp = ss.str();
    std::string filename = "output_" + timestamp + ".csv";
    std::ofstream outFile( filename );

    outFile << "Diff Bits,Prime1,Prime2\n";
    for (int i = 0; i < primeCount; ++i) {
        outFile << topHashes[i] << "," << primes[i] << "," << topPrime1[i] << "\n";
    }
    outFile.close();

    // Pause execution and wait for user input
    std::cout << "Press Enter to continue...";
    std::cin.get();

    return 0;
}
